#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

__global__ void hello (void)
{
	printf("Hello Wold from GPU!\n");
}

extern "C" int fun_cuda()
{
	int c;
	int *dev_c;
	hipMalloc((void **)&dev_c, sizeof(int));
	add<<<1,1>>>(2, 7, dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("2+7=%d\n",c);
	hipFree(dev_c);
	hello<<<1, 10>>>();
	hipDeviceReset();
	return 0;
}
